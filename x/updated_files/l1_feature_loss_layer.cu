#include <vector>

#include "caffe/layers/l1_feature_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void L1FeatureLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  int count = bottom[0]->count();
  //caffe_gpu_sub(
  //    count,
  //    bottom[0]->gpu_data(),
  //    bottom[1]->gpu_data(),
  //    diff_.mutable_gpu_data());
  
  //caffe_copy(count, bottom[0]->gpu_data(), diff_.mutable_gpu_data());
  //Dtype dot;
  //caffe_gpu_dot(count, diff_.gpu_data(), diff_.gpu_data(), &dot);
  //Dtype loss = dot / bottom[0]->num() / Dtype(2);
  //top[0]->mutable_cpu_data()[0] = loss;


  caffe_gpu_abs(count, bottom[0]->gpu_data(), temp_abs_.mutable_gpu_data());
  caffe_gpu_div(count, bottom[0]->gpu_data(), temp_abs_.gpu_data(), diff_.mutable_gpu_data());

  Dtype sum;
  caffe_gpu_asum(count, temp_abs_.gpu_data(), &sum);
  Dtype loss = sum / bottom[0]->num();
  top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
void L1FeatureLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  /*for (int i = 0; i < 2; ++i) {
    if (propagate_down[i]) {
      const Dtype sign = (i == 0) ? 1 : -1;
      const Dtype alpha = sign * top[0]->cpu_diff()[0] / bottom[i]->num();
      caffe_gpu_axpby(
          bottom[i]->count(),              // count
          alpha,                              // alpha
          diff_.gpu_data(),                   // a
          Dtype(0),                           // beta
          bottom[i]->mutable_gpu_diff());  // b
    }
  }*/
    const Dtype sign = 1;
    const Dtype alpha = sign * top[0]->cpu_diff()[0] / bottom[0]->num();
    caffe_gpu_axpby(
          bottom[0]->count(),              // count
          alpha,                              // alpha
          diff_.gpu_data(),                   // a
          Dtype(0),                           // beta
          bottom[0]->mutable_gpu_diff());  // b
    caffe_gpu_set(bottom[1]->count(), Dtype(0), bottom[1]->mutable_gpu_diff());
}

INSTANTIATE_LAYER_GPU_FUNCS(L1FeatureLossLayer);

}  // namespace caffe
