#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/l1_feature_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void shostakovich_sym_no_5(const int count, const Dtype* bottom_data, Dtype* diff_data) {
    CUDA_KERNEL_LOOP(index, count) {
        if (bottom_data[index] == 0) {
            diff_data[index] = 0;
        }
    }
}




template <typename Dtype>
void L1FeatureLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  int count = bottom[0]->count();
  //caffe_gpu_sub(
  //    count,
  //    bottom[0]->gpu_data(),
  //    bottom[1]->gpu_data(),
  //    diff_.mutable_gpu_data());
  
  //caffe_copy(count, bottom[0]->gpu_data(), diff_.mutable_gpu_data());
  //Dtype dot;
  //caffe_gpu_dot(count, diff_.gpu_data(), diff_.gpu_data(), &dot);
  //Dtype loss = dot / bottom[0]->num() / Dtype(2);
  //top[0]->mutable_cpu_data()[0] = loss;


  caffe_gpu_abs(count, bottom[0]->gpu_data(), temp_abs_.mutable_gpu_data());
  caffe_gpu_div(count, bottom[0]->gpu_data(), temp_abs_.gpu_data(), diff_.mutable_gpu_data());
  

  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* diff_data = diff_.mutable_gpu_data();

  //for (int i=0; i < bottom[0]->count(); i++) {
  //  if ((bottom_data[i]) == 0) {
  //    diff_data[i] = 0;   
  //  }
  //}
  shostakovich_sym_no_5<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
          count, bottom_data, diff_data);
  CUDA_POST_KERNEL_CHECK;



  Dtype sum;
  //caffe_gpu_asum(count, temp_abs_.gpu_data(), &sum);
  caffe_gpu_dot(count, temp_abs_.gpu_data(), uni_temp_.gpu_data(), &sum);

  Dtype loss = sum / bottom[0]->num();
  top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
void L1FeatureLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  /*for (int i = 0; i < 2; ++i) {
    if (propagate_down[i]) {
      const Dtype sign = (i == 0) ? 1 : -1;
      const Dtype alpha = sign * top[0]->cpu_diff()[0] / bottom[i]->num();
      caffe_gpu_axpby(
          bottom[i]->count(),              // count
          alpha,                              // alpha
          diff_.gpu_data(),                   // a
          Dtype(0),                           // beta
          bottom[i]->mutable_gpu_diff());  // b
    }
  }*/
    const Dtype sign = 1;
    const Dtype alpha = sign * top[0]->cpu_diff()[0] / bottom[0]->num();
    caffe_gpu_axpby(
          bottom[0]->count(),              // count
          alpha,                              // alpha
          diff_.gpu_data(),                   // a
          Dtype(0),                           // beta
          bottom[0]->mutable_gpu_diff());  // b
    caffe_gpu_set(bottom[1]->count(), Dtype(0), bottom[1]->mutable_gpu_diff());
}

INSTANTIATE_LAYER_GPU_FUNCS(L1FeatureLossLayer);

}  // namespace caffe
